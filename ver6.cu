#include "hip/hip_runtime.h"
#pragma once
#include "matmul.h"
#include <cassert>
#define CEIL_DIV(M, bkSize) ((M+bkSize-1)/bkSize)

//2D tiling
//block을 tile로 쪼개어 순회한다.
template<unsigned int blockSize, unsigned int bmSize, unsigned int bnSize, unsigned int bkSize, unsigned int TM, unsigned int TN>
__global__ void mul6(float* A, float* B, float* C, int M, int N, int K, int alpha, int beta){
    
    extern __shared__ float sharedMem[];
    float* As = sharedMem;
    float* Bs = sharedMem + blockSize * bkSize;
    
    float AsReg[TM * bkSize];
    float BsReg[bkSize * TN];
    float threadResults[TM * TN];
    for(int i=0; i<TM; i++){
        for(int j=0; j<TN; j++){
            threadResults[i*TN+j] = 0.0;
        }
    }

    // int threadCol = threadIdx.x; 
    // int threadRow = threadIdx.y;
    int threadCol = threadIdx.x % bnSize;
    int threadRow = threadIdx.x / bnSize;

    //이 블록에서 계산해야 하는 C의 원소 개수
    int numResultsPerBlock = blockSize * blockSize;
    //한 스레드에서 계산하길 원하는 C 원소 개수 (2D tiling)
    int numResultsPerThread = TM * TN;
    //블록당 스레드 개수
    int numThreadsPerBlock = numResultsPerBlock / numResultsPerThread;
    
    //한 블록은 A 행렬에서 width = bkSize, height = strideA 만큼의 영역을 한 번에 읽어온다.
    int strideA = numThreadsPerBlock / bkSize;
    //한 블록은 B 행렬에서 width = TN, height = strideB 만큼의 영역을 한 번에 읽어온다.
    int strideB = numThreadsPerBlock / bkSize;

    int innerRowA = threadIdx.x / (bkSize/4);
    int innerColA = threadIdx.x % (bkSize/4);
    int innerRowB = threadIdx.x / (bnSize/4);
    int innerColB = threadIdx.x % (bnSize/4);

    //0. 포인터를 시작점으로
    A += blockIdx.y * K * blockSize;
    B += blockIdx.x * blockSize;
    C += blockIdx.y * N * blockSize + blockIdx.x * blockSize;

    for(int i=0; i<K; i+=bkSize){

        float4 tmp = reinterpret_cast<float4 *>(&A[innerRowA * K + innerColA * 4])[0];
        As[(innerColA + 0) * bmSize + innerRowA] = tmp.x;
        As[(innerColA + 1) * bmSize + innerRowA] = tmp.y;
        As[(innerColA + 2) * bmSize + innerRowA] = tmp.z;
        As[(innerColA + 3) * bmSize + innerRowA] = tmp.w;
        
        reinterpret_cast<float4 *>(&Bs[innerRowB * bnSize + innerColB * 4])[0]
        = reinterpret_cast<float4*>(&B[innerRowB * N + innerColB * 4])[0];
        __syncthreads();
 
        A += bkSize;
        B += bkSize * N;
        float cur;

        for(int dotIdx=0; dotIdx<bkSize; ++dotIdx){
            
            //As row들 고정, Bs의 column을 바꿔가며 연산
            for(int i=0; i<TM; ++i){
                AsReg[i] = As[threadRow * TM + dotIdx * blockSize + i];
            }
            for(int j=0; j<TN; ++j){
                BsReg[j] = Bs[threadCol * TN + dotIdx * blockSize + j];
            }

            for(int r=0; r<TM; r++){
                cur = AsReg[r];
                for(int c=0; c<TN; c++){
                    threadResults[r * TN + c] += cur*BsReg[c];
                }
            }
        }
        __syncthreads();
    }

    //global 메모리에 옮겨적기
    for(int r=0; r<TM; r++){
        for(int c=0; c<TN; c+=4){
            float4 tmp = reinterpret_cast<float4*>(&C[(threadRow*TM+r)*N + threadCol*TN + c])[0];
            tmp.x = alpha * threadResults[r*TN + c + 0] + beta * tmp.x;
            tmp.y = alpha * threadResults[r*TN + c + 1] + beta * tmp.y;
            tmp.z = alpha * threadResults[r*TN + c + 2] + beta * tmp.z;
            tmp.w = alpha * threadResults[r*TN + c + 3] + beta * tmp.w;
            reinterpret_cast<float4*>(&C[(threadRow*TM + r)*N + threadCol * TN + c])[0] = tmp;
        }
    }
}

void mul66(float*A, float* B, int M, int N, int K, int alpha, int beta){
    const int TM = 4;
    const int TN = 8;
    const int blockSize = 64;

    assert(blockSize % TM == 0);
    assert(blockSize % TN ==0);

    const int bkSize = blockSize / TM;
    const int bnSize = blockSize / TN;
    const int bmSize = blockSize / TN;
    dim3 gridDim = dim3(CEIL_DIV(N, blockSize), CEIL_DIV(M,blockSize));
    dim3 blockDim = dim3(bnSize * bkSize);

    //Device memory로 값 복사
    matmul_memcpy_toDevice(A, B, M, N, K);

    //연산 수행
    mul6<blockSize, bmSize, bnSize, bkSize, TM, TN><<<gridDim, blockDim, 2*sizeof(float)*blockSize*bkSize>>>(A_gpu, B_gpu, C_gpu, M, N, K, alpha, beta);

    //Host memory로 정답 복사
    matmul_memcpy_toHost(M, N);
}