#include "matmul.h"

void matmul_init(int M, int N, int K){
    //device memory 할당
    CHECK_CUDA( hipMalloc((void**)&A_gpu, sizeof(float)*M*K) );
    CHECK_CUDA( hipMalloc((void**)&B_gpu, sizeof(float)*K*N) );
    CHECK_CUDA( hipMalloc((void**)&C_gpu, sizeof(float)*M*N) );
    C_cpu = (float*)malloc(sizeof(float)*M*N);

    CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_memcpy_toDevice(float* A, float* B, int M, int N, int K){
    CHECK_CUDA( hipMemcpy(A_gpu, A, sizeof(float)*M*K, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(B_gpu, B, sizeof(float)*K*N, hipMemcpyHostToDevice) );
}

void matmul_memcpy_toHost(int M, int N){
    CHECK_CUDA( hipMemcpy(C_cpu, C_gpu, sizeof(float)*M*N, hipMemcpyDeviceToHost) );
}

void matmul_cleanup(){
    hipFree(A_gpu);
    hipFree(B_gpu);
    hipFree(C_cpu);
    CHECK_CUDA( hipDeviceSynchronize() );
}