#include "hip/hip_runtime.h"
#pragma once
#include "matmul.h"
#include <cassert>
#define CEIL_DIV(M, bkSize) ((M+bkSize-1)/bkSize)

//2D tiling
//block을 tile로 쪼개어 순회한다.
template<unsigned int bmSize, unsigned int bnSize, unsigned int bkSize, unsigned int TM, unsigned int TN>
__global__ void mul5(float* A, float* B, float* C, int M, int N, int K, int alpha, int beta){
    
    extern __shared__ float sharedMem[];
    float* As = sharedMem;
    float* Bs = sharedMem + 32 * bkSize;
    
    float AsReg[TM * bkSize];
    float BsReg[bkSize * TN];
    float threadResults[TM * TN];
    for(int i=0; i<TM; i++){
        for(int j=0; j<TN; j++){
            threadResults[i*TN+j] = 0.0;
        }
    }

    int threadCol = threadIdx.y; // (bnSize / TN);
    int threadRow = threadIdx.x; // (bnSize / TN);

    //이 블록에서 계산해야 하는 C의 원소 개수
    int numResultsPerBlock = 32 * 32;
    //한 스레드에서 계산하길 원하는 C 원소 개수 (2D tiling)
    int numResultsPerThread = TM * TN;
    //블록당 스레드 개수
    int numThreadsPerBlock = numResultsPerBlock / numResultsPerThread;
    
    //한 블록은 A 행렬에서 width = bkSize, height = strideA 만큼의 영역을 한 번에 읽어온다.
    int strideA = numThreadsPerBlock / bkSize;
    //한 블록은 B 행렬에서 width = TN, height = strideB 만큼의 영역을 한 번에 읽어온다.
    int strideB = numThreadsPerBlock / bkSize;

    int innerRowA = threadIdx.x; // / bkSize;
    int innerColA = threadIdx.y; // % bkSize;
    int innerRowB = threadIdx.y; // / bnSize;
    int innerColB = threadIdx.x; // % bnSize;

    //0. 포인터를 시작점으로
    A += blockIdx.y * K * 32;
    B += blockIdx.x * 32;
    C += blockIdx.y * N * 32 + blockIdx.x * 32;

    for(int i=0; i<K; i+=bkSize){
        for(int j=0; j<32; j+=strideA){
           As[(innerRowA + j) * bkSize + innerColA] = A[(innerRowA + j) * K + innerColA]; 
        }
        for(int j=0; j<32; j+=strideB){
            Bs[(innerRowB)*32 + innerColB + j] = B[(innerRowB)*N + innerColB + j];
        }
        __syncthreads();
 
        A += bkSize;
        B += bkSize * N;
        float cur;

        for(int dotIdx=0; dotIdx<bkSize; ++dotIdx){
            
            //As row들 고정, Bs의 column을 바꿔가며 연산
            for(int i=0; i<TM; ++i){
                AsReg[i] = As[threadCol * TM * bkSize + i*bkSize + dotIdx];
            }
            for(int j=0; j<TN; ++j){
                BsReg[j] = Bs[threadRow * TN + dotIdx * 32 + j];
            }

            for(int r=0; r<TM; r++){
                cur = AsReg[r];
                for(int c=0; c<TN; c++){
                    threadResults[r * TN + c] += cur*BsReg[c];
                }
            }
        }
        __syncthreads();
    }

    //global 메모리에 옮겨적기
    for(int r=0; r<TM; r++){
        for(int c=0; c<TN; c++){
            C[(threadCol * TM + r) * N + threadRow * TN + c] = alpha * threadResults[r*TN + c] + beta * C[(threadCol * TM + r) * N + threadRow * TN + c];
        }
    }
}

void mul55(float*A, float* B, int M, int N, int K, int alpha, int beta){
    const int TM = 4;
    const int TN = 16;
    const int blockSize = 32;

    assert(blockSize % TM == 0);
    assert(blockSize % TN ==0);

    const int bkSize = blockSize / TM;
    const int bnSize = blockSize / TN;
    const int bmSize = blockSize / TN;
    dim3 gridDim = dim3(CEIL_DIV(N, blockSize), CEIL_DIV(M,blockSize));
    dim3 blockDim = dim3(bnSize, bkSize);

    //Device memory로 값 복사
    matmul_memcpy_toDevice(A, B, M, N, K);

    printf("mul55\n");
    printf("bm : %d, bn : %d, bk : %d, TM : %d, TN : %d\n",bmSize, bnSize, bkSize, TM, TN);
    //연산 수행
    mul5<bmSize, bnSize, bkSize, TM, TN><<<gridDim, blockDim, 2*sizeof(float)*32*bkSize>>>(A_gpu, B_gpu, C_gpu, M, N, K, alpha, beta);

    //Host memory로 정답 복사
    matmul_memcpy_toHost(M, N);
}