#include "hip/hip_runtime.h"
#pragma once
#include "matmul.h"
#include <cassert>
#define CEIL_DIV(M, bkSize) ((M+bkSize-1)/bkSize)

//2D tiling
//block을 tile로 쪼개어 순회한다.
template<unsigned int bmSize, unsigned int bnSize, unsigned int bkSize, unsigned int TM, unsigned int TN>
__global__ void mul5(float* A, float* B, float* C, int M, int N, int K, int alpha, int beta){
    
    extern __shared__ float sharedMem[];
    float* As = sharedMem;
    float* Bs = sharedMem + 32 * bkSize;
    
    float AsReg[TM * bkSize];
    float BsReg[bkSize * TN];
    float threadResults[TM * TN];
    for(int i=0; i<TM; i++){
        for(int j=0; j<TN; j++){
            threadResults[i*TM+j] = 0.0;
        }
    }

    int threadCol = threadIdx.y; // (bnSize / TN);
    int threadRow = threadIdx.x; // (bnSize / TN);

    //이 블록에서 계산해야 하는 C의 원소 개수
    int numResultsPerBlock = 32 * 32;
    //한 스레드에서 계산하길 원하는 C 원소 개수 (2D tiling)
    int numResultsPerThread = TM * TN;
    //블록당 스레드 개수
    int numThreadsPerBlock = numResultsPerBlock / numResultsPerThread;
    
    //한 블록은 A 행렬에서 width = bkSize, height = strideA 만큼의 영역을 한 번에 읽어온다.
    int strideA = numThreadsPerBlock / bkSize;
    //한 블록은 B 행렬에서 width = bnSize, height = strideB 만큼의 영역을 한 번에 읽어온다.
    int strideB = numThreadsPerBlock / bnSize;

    int innerRowA = threadIdx.x; // / bkSize;
    int innerColA = threadIdx.y; // % bkSize;
    int innerRowB = threadIdx.x; // / bnSize;
    int innerColB = threadIdx.y; // % bnSize;

    //0. 포인터를 시작점으로
    A += blockIdx.y * K * 32;
    B += blockIdx.x * 32;
    C += blockIdx.y * N * 32 + blockIdx.x * 32;

    for(int i=0; i<K; i+=bkSize){
        for(int j=0; j<32; j+=strideA){
           As[(innerRowA + j) * bkSize + innerColA] = A[(innerRowA + j) * K + innerColA]; 
        }
        for(int j=0; j<32; j+=strideB){
            Bs[(innerRowB)*32 + innerColB + j] = B[(innerRowB)*N + innerColB + j];
        }
        __syncthreads();

        // if(blockIdx.x==0 && blockIdx.y ==0 && threadCol==2 && threadRow ==1){
        //     for(int i=0; i<bkSize; i++){
        //         for(int j=0; j<32; j++){
        //             printf("%.1f ",Bs[i*32+j]);
        //         }
        //         printf("\n");
        //     }
        //     printf("-------\n");
        // }
 
        A += bkSize;
        B += bkSize * N;
        float cur;

        for(int dotIdx=0; dotIdx<bkSize; ++dotIdx){
            
            //As row들 고정, Bs의 column을 바꿔가며 연산
            for(int i=0; i<TM; ++i){
                AsReg[i] = As[threadRow * TM * bkSize + i*bkSize + dotIdx];
            }
            for(int j=0; j<TN; ++j){
                BsReg[j] = Bs[threadCol * TN + dotIdx * 32 + j];
                if(blockIdx.x==0 && blockIdx.y ==0 && threadCol==2 && threadRow ==1){
                    //printf("BsReg[%d] = Bs[%d] = %.1f\n",j, threadCol*TN + dotIdx * 32 + j,Bs[threadCol * TN + dotIdx * 32 + j]);
                }
            }

            // if(blockIdx.x==0 && blockIdx.y ==0 && threadCol==2 && threadRow ==1){
            //     printf("dot Idx : %d\n",dotIdx);
            //     for(int i=0; i<TM; i++) printf("%.1f ",AsReg[i]);
            //     printf("\n");
            //     for(int j=0; j<TN; j++) printf("%.1f",BsReg[j]);
            //     printf("\n");
            // }

            for(int r=0; r<TM; r++){
                cur = AsReg[r];
                for(int c=0; c<TN; c++){
                    threadResults[r * TN + c] += cur*BsReg[c];
                }
            }
        }
        __syncthreads();
    }

    //global 메모리에 옮겨적기
    for(int r=0; r<TM; r++){
        for(int c=0; c<TN; c++){
            C[(threadRow * TM + r) * N + threadCol * TN + c] = alpha * threadResults[r*TN + c] + beta * C[(threadRow * TM + r) * N + threadCol * TN + c];
            //if(threadCol==2 && threadRow==1) printf("C[%d] : %.1f\n",(threadRow * TM + r) * N + threadCol * TN + c, C[(threadRow * TM + r) * N + threadCol * TN + c]);
        }
    }
}

void mul55(float*A, float* B, int M, int N, int K, int alpha, int beta){
    const int TM = 8;
    const int TN = 8;
    const int blockSize = 32;

    assert(blockSize % TM == 0);
    assert(blockSize % TN ==0);

    const int bkSize = blockSize / TM;
    const int bnSize = blockSize / TN;
    const int bmSize = blockSize / TN;
    dim3 gridDim = dim3(CEIL_DIV(N, blockSize), CEIL_DIV(M,blockSize));
    dim3 blockDim = dim3(bnSize, bkSize);

    //Device memory로 값 복사
    matmul_memcpy_toDevice(A, B, M, N, K);

    printf("mul55\n");
    printf("bm : %d, bn : %d, bk : %d, TM : %d, TN : %d\n",bmSize, bnSize, bkSize, TM, TN);
    //연산 수행
    mul5<bmSize, bnSize, bkSize, TM, TN><<<gridDim, blockDim, 2*sizeof(float)*32*bkSize>>>(A_gpu, B_gpu, C_gpu, M, N, K, alpha, beta);

    //Host memory로 정답 복사
    matmul_memcpy_toHost(M, N);
}